#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>

#include "cuda_img.h"

__global__ void kernel_cp(CudaImg bg_image, CudaImg fg_image, int2 bg_pos, int2 pos, int2 pos2)
{
    int2 fg_index = {(int)threadIdx.x + blockIdx.x * blockDim.x, (int)threadIdx.y + blockIdx.y * blockDim.y};

    if (fg_index.x >= fg_image.m_size.x)
        return;

    if (fg_index.y >= fg_image.m_size.y)
        return;

    if (fg_index.x < pos.x || fg_index.x > pos2.x)
        return;

    if (fg_index.y < pos.y || fg_index.y > pos2.y)
        return;

    int2 bg_index = {fg_index.x + bg_pos.x, fg_index.y + bg_pos.y};

    if (bg_index.x < 0 || bg_index.x >= bg_image.m_size.x)
        return;

    if (bg_index.y < 0 || bg_index.y >= bg_image.m_size.y)
        return;

    bg_image.at<uchar3>(bg_index.x, bg_index.y) = fg_image.at<uchar3>(fg_index.x, fg_index.y);
}

void cu_cp(CudaImg bg_image, CudaImg fg_image, int2 bg_pos, int2 pos, int2 pos2)
{
    hipError_t l_cerr;

    int block_size = 32;
    dim3 blocks(((fg_image.m_size.x + block_size - 1) / block_size), ((fg_image.m_size.y + block_size - 1) / block_size));
    dim3 threads(block_size, block_size);

    kernel_cp<<<blocks, threads>>>(bg_image, fg_image, bg_pos, pos, pos2);

    if ((l_cerr = hipGetLastError()) != hipSuccess)
        printf("CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString(l_cerr));

    hipDeviceSynchronize();
}

__global__ void kernel_resize(CudaImg image, CudaImg copy)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < copy.m_size.x && y < copy.m_size.y)
    {
        float u = (float)x * image.m_size.x / copy.m_size.x;
        float v = (float)y * image.m_size.y / copy.m_size.y;

        int x1 = (int)floorf(u);
        int y1 = (int)floorf(v);
        int x2 = x1 + 1;
        int y2 = y1 + 1;

        float weight1 = (x2 - u) * (y2 - v);
        float weight2 = (u - x1) * (y2 - v);
        float weight3 = (x2 - u) * (v - y1);
        float weight4 = (u - x1) * (v - y1);

        copy.at<uchar3>(x, y).x = static_cast<uchar>(weight1 * image.at<uchar3>(x1, y1).x + weight2 * image.at<uchar3>(x1, y2).x + weight3 * image.at<uchar3>(x2, y1).x + weight4 * image.at<uchar3>(x2, y2).x);
        copy.at<uchar3>(x, y).y = static_cast<uchar>(weight1 * image.at<uchar3>(x1, y1).y + weight2 * image.at<uchar3>(x1, y2).y + weight3 * image.at<uchar3>(x2, y1).y + weight4 * image.at<uchar3>(x2, y2).y);
        copy.at<uchar3>(x, y).z = static_cast<uchar>(weight1 * image.at<uchar3>(x1, y1).z + weight2 * image.at<uchar3>(x1, y2).z + weight3 * image.at<uchar3>(x2, y1).z + weight4 * image.at<uchar3>(x2, y2).z);
    }
}

void cu_resize(CudaImg image, CudaImg copy)
{
    hipError_t l_cerr;

    int block_size = 32;
    dim3 blocks(((copy.m_size.x + block_size - 1) / block_size), ((copy.m_size.y + block_size - 1) / block_size));
    dim3 threads(block_size, block_size);

    kernel_resize<<<blocks, threads>>>(image, copy);

    if ((l_cerr = hipGetLastError()) != hipSuccess)
        printf("CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString(l_cerr));

    hipDeviceSynchronize();
}

__global__ void kernel_rotate(CudaImg image, CudaImg copy)
{
    int2 index = {(int)threadIdx.x + blockIdx.x * blockDim.x, (int)threadIdx.y + blockIdx.y * blockDim.y};

    if (index.x >= image.m_size.x)
        return;

    if (index.y >= image.m_size.y)
        return;

    copy.at<uchar3>(copy.m_size.y - index.y - 1, index.x) = image.at<uchar3>(index.x, index.y);
}

void cu_rotate(CudaImg image, CudaImg copy)
{
    hipError_t l_cerr;

    int block_size = 32;
    dim3 blocks(((copy.m_size.x + block_size - 1) / block_size), ((copy.m_size.y + block_size - 1) / block_size));
    dim3 threads(block_size, block_size);

    kernel_rotate<<<blocks, threads>>>(image, copy);

    if ((l_cerr = hipGetLastError()) != hipSuccess)
        printf("CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString(l_cerr));

    hipDeviceSynchronize();
}