#include "hip/hip_runtime.h"
// ***********************************************************************
//
// Demo program for education in subject
// Computer Architectures and Parallel Systems.
// Petr Olivka, dep. of Computer Science, FEI, VSB-TU Ostrava, 2020/11
// email:petr.olivka@vsb.cz
//
// Example of CUDA Technology Usage with unified memory.
//
// Image transformation from RGB to BW schema.
//
// ***********************************************************************

#include <stdio.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>

#include "cuda_img.h"

__global__ void kernel_polovina(CudaImg image)
{
    int index_x = threadIdx.x + blockIdx.x * blockDim.x;
    int index_y = threadIdx.y + blockIdx.y * blockDim.y;

    if (index_x >= image.m_size.x)
        return;

    if (index_y >= image.m_size.y)
        return;

    image.at<uchar3>(index_y, index_x).x /= 2;
    image.at<uchar3>(index_y, index_x).y /= 2;
    image.at<uchar3>(index_y, index_x).z /= 2;
}

void cu_polovina(CudaImg image)
{
    hipError_t l_cerr;

    int block_size = 32;
    dim3 blocks(((image.m_size.x + block_size - 1) / block_size), ((image.m_size.y + block_size - 1) / block_size));
    dim3 threads(block_size, block_size);

    kernel_polovina<<<blocks, threads>>>(image);

    if ((l_cerr = hipGetLastError()) != hipSuccess)
        printf("CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString(l_cerr));

    hipDeviceSynchronize();
}

__global__ void kernel_bloky(CudaImg image, int x)
{
    int index_x = threadIdx.x + blockIdx.x * blockDim.x;
    int index_y = threadIdx.y + blockIdx.y * blockDim.y;

    if (index_x >= image.m_size.x)
        return;

    if (index_y >= image.m_size.y)
        return;

    int blok = (blockIdx.x + blockIdx.y + x) % 3;

    if (blok == 0)
    {
        image.at<uchar3>(index_y, index_x).z = 0;
        image.at<uchar3>(index_y, index_x).y = 0;
    }

    if (blok == 1)
    {
        image.at<uchar3>(index_y, index_x).z = 0;
        image.at<uchar3>(index_y, index_x).x = 0;
    }

    if (blok == 2)
    {
        image.at<uchar3>(index_y, index_x).y = 0;
        image.at<uchar3>(index_y, index_x).x = 0;
    }
}

void cu_bloky(CudaImg image, int x)
{
    hipError_t l_cerr;

    int block_size = 32;
    dim3 blocks(((image.m_size.x + block_size - 1) / block_size), ((image.m_size.y + block_size - 1) / block_size));
    dim3 threads(block_size, block_size);

    kernel_bloky<<<blocks, threads>>>(image, x);

    if ((l_cerr = hipGetLastError()) != hipSuccess)
        printf("CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString(l_cerr));

    hipDeviceSynchronize();
}